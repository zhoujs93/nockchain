
#include <hip/hip_runtime.h>
extern "C" __global__
void ntt_forward(unsigned long long* buf, unsigned int n) {
    // TODO: implement butterfly steps over your field (e.g., Goldilocks) with twiddles in __constant__ memory.
    // For now, no-op copy style kernel (does nothing).
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) { buf[i] = buf[i]; }
}

extern "C" __global__
void ntt_inverse(unsigned long long* buf, unsigned int n) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) { buf[i] = buf[i]; }
}
